#include "hip/hip_runtime.h"
/*
� Marcin Wodejko 2024.
marwod@interia.pl
*/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include ""
#include <hip/hip_texture_types.h>
#include <stdint.h>
#include <string>
#include <time.h>
#include <iostream>
#include <cmath>
#include <conio.h>
#include <math.h>
#include <cmath>
#include <vector>
#include "opencv2/opencv.hpp"
#include "opencv2/core.hpp"
#include "Naglowek_struktury.h"
#include "transformaty_cuda.cuh"
#include <random>
#include <cstdlib>
#include <filesystem> // musi byc C++17 lub wyzej


//.werrsja po poprawkach 8 maja 2024, bardzo szybka. poprawiono transformate walsha


#define ROZMIAR_OBSZARU_PRZESZUKANIA       32 //warto�� w ilo�ci �atek i u�ywanych w�tk�w. rozmiar w pixelax wyniesie 40 (po dodaniu rozmiaru �atki
#define ROZMIAR_LATKI       8
#define POWIERZCHNIA_LATKI       64
#define RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA 40 // ROZMIAR_PRZESZUKANIA +ROZMIAR_�ATKI ilo�� pixeli obszaru przeszukania
#define SIGMA       20.0f
#define LAMBDA2DHARD       0.9f
#define P_HARD  3 //p_Hard krok tworzenia latek, w oryginale 1,2 lub 3, u Lebruna 3
#define P_WIEN  3 //krok tworzenia latek, w oryginale 1, 2 lub 3, u Lebruna 3
#define N_HARD  16 //maks ilosc lek w grupie 3D
#define N_WIEN  32 //maks ilosc lek w grupie 3D
#define TAU_HARD_NISKI 4000.0 //maksymalna odleglo�c MSE latki przysz szumie niskim
#define TAU_HARD_WYSOKI 3000.0 //maksymalna odleglo�c MSE latki przysz szumie niskim
#define LAMBDA3D_HARD 2.7  //LambdaHard2d	progowanie(trasholding) Grupy3d w pierwszym kroku filtra, u Lebruna 2,7
#define LAMBDA2D_HARD 2.0//Lambda_hard3d progowanie(trasholding) przy block matchingu, u Lebruna 2.0




__constant__ float Macierz_wspolczynnikow_Kaizerra[POWIERZCHNIA_LATKI] = 
{ 0.1924, 0.2989, 0.3846, 0.4325, 0.4325, 0.3846, 0.2989, 0.1924,
  0.2989, 0.4642, 0.5974, 0.6717, 0.6717, 0.5974, 0.4642, 0.2989,
  0.3846, 0.5974, 0.7688, 0.8644, 0.8644, 0.7688, 0.5974, 0.3846,
  0.4325, 0.6717, 0.8644, 0.9718, 0.9718, 0.8644, 0.6717, 0.4325,
  0.4325, 0.6717, 0.8644, 0.9718, 0.9718, 0.8644, 0.6717, 0.4325,
  0.3846, 0.5974, 0.7688, 0.8644, 0.8644, 0.7688, 0.5974, 0.3846,
  0.2989, 0.4642, 0.5974, 0.6717, 0.6717, 0.5974, 0.4642, 0.2989,
  0.1924, 0.2989, 0.3846, 0.4325, 0.4325, 0.3846, 0.2989, 0.1924 };

__constant__ float  aConst_macierz_wspolczynnikow2d_1[POWIERZCHNIA_LATKI] =
{ 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536, 0.3536,
0.4904, 0.4157, 0.2778, 0.0976, -0.0975, -0.2778, -0.4157, -0.4904,
0.4619, 0.1913, -0.1913, -0.462, -0.462, -0.1913, 0.1913, 0.4619,
0.4157, -0.0975, -0.4904, -0.2778, 0.2778, 0.4904, 0.0976, -0.4157,
0.3536, -0.3536, -0.3536, 0.3536, 0.3536, -0.3536, -0.3536, 0.3536 ,
0.2778, -0.4904, 0.0975, 0.4157, -0.4157, -0.0975, 0.4904, -0.2778,
0.1913, -0.4619, 0.4619, -0.1913, -0.1913, 0.4619, -0.4619, 0.1913,
0.0975, -0.2778, 0.4157, -0.4904, 0.4904, -0.4157, 0.2778, -0.0975};

__constant__ float aConst_macierz_wspolczynnikow2d_2[POWIERZCHNIA_LATKI] =
{ 0.3536, 0.4904, 0.4619, 0.4157, 0.3536, 0.2778, 0.1914, 0.0975,  
  0.3536, 0.4157, 0.1914, -0.0975, -0.3536, -0.4904, -0.4619, -0.2778,
  0.3536, 0.2778, -0.1914, -0.4904, -0.3536, 0.0975, 0.4619, 0.4157,
  0.3536, 0.0976, -0.462, -0.2778, 0.3536, 0.4157, -0.1913, -0.4904,
  0.3536, -0.0976, -0.462, 0.2778, 0.3536, -0.4157, -0.1915, 0.4904,
  0.3536, -0.2778, -0.1914, 0.4904, -0.3535, -0.0977, 0.4620, -0.4157,
  0.3536, -0.4157, 0.1913, 0.0977, -0.3536, 0.4904, -0.4619, 0.2778, 
  0.3536, -0.4904, 0.4619, -0.4157, 0.3534, -0.2778, 0.1911, -0.0975 };


__global__ void Najmniejsze_liczby(Tablice_koordynatLatek koordynatySOA, int* device_tablica_ilosci_pasujacych_latek, int ilosc_najmniejszych, int tau_hard, bool krok2) // wykorzystanie algorytmu redykcji u�ywanego zwykle do sumowania tablicy
{

    int indeks = threadIdx.x;

    int przesuniecie = blockIdx.z * ROZMIAR_OBSZARU_PRZESZUKANIA * ROZMIAR_OBSZARU_PRZESZUKANIA;
    __shared__ int s_koordynatySOA[1024];
    __shared__ float s_MSE_SOA[1024];
    __shared__ int s_koordynaty_najmniejszych_SOA[N_WIEN];
    for (int i = 0; i < 2; i++)
    {
        if (indeks < 512)
        {
            s_MSE_SOA[indeks + (i * 512)] = koordynatySOA.MSE[przesuniecie + indeks + (i * 512)];
            s_koordynatySOA[indeks + (i * 512)] = indeks + (i * 512);

        }
    }
    __syncthreads();

    for (int i = 0; i < ilosc_najmniejszych; i++)//ilosc najmniejszych wynosi N_HARD dla pierwszego kroku lub 32 dla drugiego st�d w pami�ci zarezerwowano miejsce dla 32
    {

        for (int s = 512; s > 0; s >>= 1)//s zmniejszamy dwukrotnie za ka�d� iteracj�
        {

            if (threadIdx.x < s)
            {
                if (s_MSE_SOA[s_koordynatySOA[indeks]] > s_MSE_SOA[s_koordynatySOA[indeks + s]])
                {
                    s_koordynatySOA[indeks] = s_koordynatySOA[indeks + s];
                }
            }
        }
        //__syncthreads();

        if (threadIdx.x == 0)
        {
            s_koordynaty_najmniejszych_SOA[i] = s_koordynatySOA[0];
            s_MSE_SOA[s_koordynatySOA[0]] = 10000000000000000000; //aby w nastepnej iteracji wyszukiwania zosta� zigm=norowana jako kandydat do najmniejszego
        }
    }
    //__syncthreads();

    if (indeks < ilosc_najmniejszych)
    {
        float tymczasowa_MSE = koordynatySOA.MSE[przesuniecie + s_koordynaty_najmniejszych_SOA[indeks]];
        int tymczasowa_x = koordynatySOA.koordynata_x[przesuniecie + s_koordynaty_najmniejszych_SOA[indeks]];
        int tymczasowa_y = koordynatySOA.koordynata_y[przesuniecie + s_koordynaty_najmniejszych_SOA[indeks]];

        koordynatySOA.MSE[indeks + przesuniecie] = tymczasowa_MSE;
        koordynatySOA.koordynata_x[indeks + przesuniecie] = tymczasowa_x;
        koordynatySOA.koordynata_y[indeks + przesuniecie] = tymczasowa_y;
    }
    __syncthreads();

    if (indeks < 1)
    {

        for (int i = ilosc_najmniejszych; i > 0; i = i / 2)
        {

            if (koordynatySOA.MSE[przesuniecie + i - 1] < tau_hard)
            {
                device_tablica_ilosci_pasujacych_latek[blockIdx.z] = i;
                break;
            }
        }

        if (device_tablica_ilosci_pasujacych_latek[blockIdx.z] == 0)
        {
            device_tablica_ilosci_pasujacych_latek[blockIdx.z] = 1;
        }

    }

}


__global__ void Kalkulator_MSE(float* __restrict__ device_obrazek_poczatkowy, Tablice_koordynatLatek dev_koordynatySOA, int ilosc_blokow_w_boku_x, int ilosc_blokow_w_boku_y, int szerokosc, int i, int j)
{

    int row_pos = threadIdx.y;
    int col_pos = threadIdx.x;
    int index_x_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z % ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + i;
    int index_y_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z / ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + j;
    int index2dObszaru = threadIdx.x + threadIdx.y * ROZMIAR_OBSZARU_PRZESZUKANIA;
    int index_elmentu_zero_tablicy_koordynat = (blockIdx.z * ROZMIAR_OBSZARU_PRZESZUKANIA * ROZMIAR_OBSZARU_PRZESZUKANIA);
    __shared__ float latka_referencyjna[POWIERZCHNIA_LATKI];
    __shared__ float obszar_preszukana_shared[(ROZMIAR_OBSZARU_PRZESZUKANIA + ROZMIAR_LATKI) * (ROZMIAR_OBSZARU_PRZESZUKANIA + ROZMIAR_LATKI)];
    int ofset = RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 2;
    if ((row_pos < RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 2) && (col_pos < RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 2))
    {

        for (int i = 0; i < 2; i++)
        {
            for (int j = 0; j < 2; j++)
            {
                obszar_preszukana_shared[(row_pos + i * ofset) * (RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA)+col_pos + (j * ofset)] = (device_obrazek_poczatkowy[((row_pos + i * ofset) + index_y_pixela_gorny_lewy_obszaru_przeszukania) * szerokosc + col_pos + (j * ofset) + index_x_pixela_gorny_lewy_obszaru_przeszukania]);
                //przpisujemy obszar preszukania (40 pixeli) dla �atki do pami�ci dzielonej bloku, ze wzgl�du na zmieszczenie si� w dost�pnej w wywo�aniu funkcji iosci w�tk�w musia�em zrealizowa� przypisanie w czterech krokach.
            }

        }
    }
    __syncthreads();
    if ((row_pos < ROZMIAR_LATKI) && (col_pos < ROZMIAR_LATKI)) //przpisujemy obszar �atki do ktorej b�dziemy porownywa� do pami�ci dzielonej bloku
    {
        latka_referencyjna[row_pos * ROZMIAR_LATKI + col_pos] = obszar_preszukana_shared[(row_pos + ofset) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + (col_pos + ofset)];
        // przypisujemy warto�ci dla �atki referencyjnej (latka o rozmiarze 8*8) dla ka�dego obszaru przeszukania. Ofset jest potrzebny w zwi�zku z r�n� wielko�cia �atki i obaszaru przeszukania oraz tym �e �atka referencyjna umieszona jest w �rodku tzn jej lewy gorny r�g jest umieszczony w �rodku x=19, y=19.
        //�atka nie jest umieszczona idealnie po �rodu, ale jest to kompromis kt�ry zapewnia pokrycie ca�go obszaru i wsp�prac� z 1024 w�tkami w bloku.
    }
    __syncthreads();

    if ((row_pos < (ROZMIAR_OBSZARU_PRZESZUKANIA)) && (col_pos < (ROZMIAR_OBSZARU_PRZESZUKANIA))) //Obliczamy MSE dla 32*32 latek z obszaru przeszukania
    {
        dev_koordynatySOA.koordynata_x[index_elmentu_zero_tablicy_koordynat + index2dObszaru] = col_pos;
        dev_koordynatySOA.koordynata_y[index_elmentu_zero_tablicy_koordynat + index2dObszaru] = row_pos;
        float MSE = 0;

        for (int i = 0; i < ROZMIAR_LATKI; i++)
        {
            for (int j = 0; j < ROZMIAR_LATKI; j++)
            {

                MSE += (((latka_referencyjna[i * ROZMIAR_LATKI + j] - obszar_preszukana_shared[(row_pos + i) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + col_pos + j])) * (latka_referencyjna[i * ROZMIAR_LATKI + j] - obszar_preszukana_shared[(row_pos + i) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + col_pos + j]));
                //__syncthreads(); // �eby nie by�o pchania si� w�tk�w jednocze�nie do tech samych pikseli slre niestety nie dzia�� przez p�tl� jhe�li wszystkie w�tki nie s� zatrudnione
            }
           
        }
        __syncthreads();
        dev_koordynatySOA.MSE[index_elmentu_zero_tablicy_koordynat + index2dObszaru] = (MSE / (POWIERZCHNIA_LATKI));
    }
}

__global__ void Kalkulator_MSE_szum_duzy(float* __restrict__ Obrazek, Tablice_koordynatLatek dev_koordynatySOA, int ilosc_blokow_w_boku_x, int szerokosc, int i, int j, int sigma)
{

    int index_x_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z % ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + i;
    int index_y_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z / ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + j;
    int index2dLatki = threadIdx.x + threadIdx.y * ROZMIAR_LATKI;
    int index_elmentu_zero_tablicy_koordynat = (blockIdx.z * ROZMIAR_OBSZARU_PRZESZUKANIA * ROZMIAR_OBSZARU_PRZESZUKANIA);
    int ofset = RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 2;
    __shared__ float  sz_Const_macierz_wspolczynnikow2d_1[POWIERZCHNIA_LATKI+8];
    __shared__ float sz_Const_macierz_wspolczynnikow2d_2[POWIERZCHNIA_LATKI+8];
 
    __shared__ float latka_referencyjna[ROZMIAR_LATKI * ROZMIAR_LATKI];
    __shared__ float latka_porownywana[ROZMIAR_LATKI * ROZMIAR_LATKI];
    __shared__ float Macierz_wynikowa_posrednia[POWIERZCHNIA_LATKI];

    if ((threadIdx.y < ROZMIAR_LATKI) && (threadIdx.x < ROZMIAR_LATKI))
    {

        sz_Const_macierz_wspolczynnikow2d_1[(threadIdx.y * (ROZMIAR_LATKI+1) + threadIdx.x)] = aConst_macierz_wspolczynnikow2d_1[threadIdx.y * ROZMIAR_LATKI + threadIdx.x];
        sz_Const_macierz_wspolczynnikow2d_2[(threadIdx.y * (ROZMIAR_LATKI+1) + threadIdx.x)] = aConst_macierz_wspolczynnikow2d_2[threadIdx.y * ROZMIAR_LATKI + threadIdx.x];
        
        //sz_Const_macierz_wspolczynnikow2d_1[threadIdx.y] [threadIdx.x] = aConst_macierz_wspolczynnikow2d_1[threadIdx.y * ROZMIAR_LATKI + threadIdx.x];
        //sz_Const_macierz_wspolczynnikow2d_2[threadIdx.y][threadIdx.x] = aConst_macierz_wspolczynnikow2d_2[threadIdx.y * ROZMIAR_LATKI + threadIdx.x];

        latka_referencyjna[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] = Obrazek[((threadIdx.y + ofset - 4) + index_y_pixela_gorny_lewy_obszaru_przeszukania) * szerokosc + threadIdx.x + (ofset - 4) + index_x_pixela_gorny_lewy_obszaru_przeszukania];
        //__syncthreads();

        latka_porownywana[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] = Obrazek[((threadIdx.y + blockIdx.y) + index_y_pixela_gorny_lewy_obszaru_przeszukania) * szerokosc + threadIdx.x + blockIdx.x + index_x_pixela_gorny_lewy_obszaru_przeszukania];
        //__syncthreads();

    }
    __syncthreads();
    if ((threadIdx.y < ROZMIAR_LATKI) && (threadIdx.x < ROZMIAR_LATKI))
    {
        Macierz_wynikowa_posrednia[index2dLatki] = 0;
        __syncthreads();
        for (int k = 0; k < ROZMIAR_LATKI; k++)

        {
            Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += sz_Const_macierz_wspolczynnikow2d_1[(threadIdx.y * (ROZMIAR_LATKI+1) + k)] * latka_referencyjna[k * ROZMIAR_LATKI + threadIdx.x];
            //Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += sz_Const_macierz_wspolczynnikow2d_1[threadIdx.y ][k] * latka_referencyjna[k * ROZMIAR_LATKI + threadIdx.x];
            //__syncthreads();
        }
        __syncthreads();
        latka_referencyjna[index2dLatki] = 0;
        __syncthreads();
        for (int k = 0; k < ROZMIAR_LATKI; k++)

        {
            latka_referencyjna[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + k] * sz_Const_macierz_wspolczynnikow2d_2[(k * (ROZMIAR_LATKI+1) + threadIdx.x)];
            //latka_referencyjna[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + k] * sz_Const_macierz_wspolczynnikow2d_2[k] [threadIdx.x];
            //__syncthreads();
        }
    }
    __syncthreads();
    if ((threadIdx.y < ROZMIAR_LATKI) && (threadIdx.x < ROZMIAR_LATKI))
    {
        Macierz_wynikowa_posrednia[index2dLatki] = 0;
        for (int k = 0; k < ROZMIAR_LATKI; k++)

        {
            Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += sz_Const_macierz_wspolczynnikow2d_1[threadIdx.y * (ROZMIAR_LATKI+1) + k] * latka_porownywana[k * ROZMIAR_LATKI + threadIdx.x];
            //Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += sz_Const_macierz_wspolczynnikow2d_1[threadIdx.y] [k ] * latka_porownywana[k * ROZMIAR_LATKI + threadIdx.x];
            //__syncthreads();
        }
        __syncthreads();
        latka_porownywana[index2dLatki] = 0;
        __syncthreads();
        for (int k = 0; k < ROZMIAR_LATKI; k++)
        {
            latka_porownywana[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + k] * sz_Const_macierz_wspolczynnikow2d_2[k * (ROZMIAR_LATKI+1) + threadIdx.x];
            //latka_porownywana[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] += Macierz_wynikowa_posrednia[threadIdx.y * ROZMIAR_LATKI + k] * sz_Const_macierz_wspolczynnikow2d_2[k][threadIdx.x];
            //__syncthreads();
        }
    }
    __syncthreads();
    if ((threadIdx.y < ROZMIAR_LATKI) && (threadIdx.x < ROZMIAR_LATKI))
    {
        
        if (fabs(latka_referencyjna[threadIdx.y * ROZMIAR_LATKI + threadIdx.x])
            < LAMBDA2D_HARD * sigma)
        {
            latka_referencyjna[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] = 0;
        }
        if (fabs(latka_porownywana[threadIdx.y * ROZMIAR_LATKI + threadIdx.x])
            < LAMBDA2D_HARD * sigma)
        {
            latka_porownywana[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] = 0;
        }
        
        //latka_referencyjna[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] = fmaxf((latka_referencyjna[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] - LAMBDA2D_HARD * sigma),0);
        //latka_porownywana[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] = fmaxf((latka_porownywana[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] -LAMBDA2D_HARD * sigma),0);
    }
    __syncthreads();

    if ((threadIdx.y < ROZMIAR_LATKI) && (threadIdx.x < ROZMIAR_LATKI))
    {
        float zmienna_sumowana = ((latka_referencyjna[index2dLatki] - latka_porownywana[index2dLatki]) * (latka_referencyjna[index2dLatki] - latka_porownywana[index2dLatki]));
        latka_referencyjna[index2dLatki] = zmienna_sumowana;

        __syncthreads();

        if (threadIdx.x == 0 && threadIdx.y == 0)
        {
            float suma = 0;
            for (int i = 0; i < 64; i++)//atomic add dla float�w jest wolniejsze
            {
                suma = suma + latka_referencyjna[i];
            }
            dev_koordynatySOA.MSE[index_elmentu_zero_tablicy_koordynat + (blockIdx.x + blockIdx.y * 32)] = suma / POWIERZCHNIA_LATKI;
            dev_koordynatySOA.koordynata_x[index_elmentu_zero_tablicy_koordynat + blockIdx.x + blockIdx.y * 32] = blockIdx.x;
            dev_koordynatySOA.koordynata_y[index_elmentu_zero_tablicy_koordynat + blockIdx.x + blockIdx.y * 32] = blockIdx.y;
        }
    }
}

__global__ void Przepisywacz_do_tabloc_transformaty(float* __restrict__ obrazek_przepisywany, Tablice_koordynatLatek tablica_koordynat_latek_SOA, int* tablica_ilosci_pasujacych_latek, float* tablice_transformaty, int ilosc_blokow_w_boku_x, int ilosc_blokow_w_boku_y, int szerokosc, int i, int j, int mnoznik_tablicy_transormat)
{
    int row_pos = threadIdx.y;
    int col_pos = threadIdx.x;
    int index_x_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z % ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + i;
    int index_y_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z / ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + j;
    int index_elmentu_zero_tablicy_koordynat = (blockIdx.z * ROZMIAR_OBSZARU_PRZESZUKANIA * ROZMIAR_OBSZARU_PRZESZUKANIA);
    int index_elmentu_zero_tablicy_transformat = (blockIdx.z * ROZMIAR_LATKI * ROZMIAR_LATKI * mnoznik_tablicy_transormat);

    __shared__ float obszar_preszukana_shared[(ROZMIAR_OBSZARU_PRZESZUKANIA + ROZMIAR_LATKI) * (ROZMIAR_OBSZARU_PRZESZUKANIA + ROZMIAR_LATKI)];
    int ofset = RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 5;
    if ((row_pos < RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 5) && (col_pos < RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 5))
    {
        for (int i = 0; i < 5; i++)
        {
            for (int j = 0; j < 5; j++)
            {
                obszar_preszukana_shared[(row_pos + i * ofset) * (RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA)+col_pos + (j * ofset)] = obrazek_przepisywany[((row_pos + i * ofset) + index_y_pixela_gorny_lewy_obszaru_przeszukania) * szerokosc + col_pos + (j * ofset) + index_x_pixela_gorny_lewy_obszaru_przeszukania];
                //przpisujemy obszar preszukania (40 pixeli) dla �atki do pami�ci dzielonej bloku, ze wzgl�du na zmieszczenie si� w dost�pnej w wywo�aniu funkcji iosci w�tk�w musia�em zrealizowa� przypisanie w czterech krokach.
            }
        }
    }
    __syncthreads();

    if ((row_pos < ROZMIAR_LATKI) && (col_pos < ROZMIAR_LATKI))
    {
        for (int i = 0; i < tablica_ilosci_pasujacych_latek[blockIdx.z]; i++)//�atka po �atce przepsisujemy �atki z obszaru przeszukania do tablicy transformat device_tablice_transformaty_32_1krok(dla 1 kroku, dl 2 kroku device_tablice_transformaty_32_
        {
            {
                int indeks_pomocniczy1 = col_pos + (row_pos)*ROZMIAR_LATKI + (i * ROZMIAR_LATKI * ROZMIAR_LATKI);
                int indeks_pomocniczy2 = (tablica_koordynat_latek_SOA.koordynata_x[index_elmentu_zero_tablicy_koordynat + i]) + col_pos + (tablica_koordynat_latek_SOA.koordynata_y[index_elmentu_zero_tablicy_koordynat + i] + row_pos) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA;
                tablice_transformaty[index_elmentu_zero_tablicy_transformat + indeks_pomocniczy1] = obszar_preszukana_shared[indeks_pomocniczy2];
            }
        }
    }
    //__syncthreads();
}
__global__ void Przepisywacz_z_tablic_transformaty_1krok(int* tablica_ilosci_zerowan, float* obrazek_po_kolejnym_kroku, float* obrazek_po_kolejnym_kroku_dzielnik, Tablice_koordynatLatek dev_koordynatySOA, int* device_tablica_ilosci_pasujacych_latek, float* device_tablice_transformaty_32_1krok, int ilosc_blokow_w_boku_x, int ilosc_blokow_w_boku_y, int szerokosc, int i, int j, int mnoznik_tablicy_transormat)
{
    if (tablica_ilosci_zerowan[blockIdx.z] < 1)
    {
        tablica_ilosci_zerowan[blockIdx.z] = 1;
    }
    int row_pos = threadIdx.y;
    int col_pos = threadIdx.x;
    int index_x_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z % ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + i;//przetestowa� czy blo z czy y czy jeden i drugi!!!!!!!
    int index_y_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z / ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + j;
    int index_2d_latki = col_pos + (row_pos * ROZMIAR_LATKI);
    int index_elmentu_zero_tablicy_koordynat = (blockIdx.z * ROZMIAR_OBSZARU_PRZESZUKANIA * ROZMIAR_OBSZARU_PRZESZUKANIA);
    int index_elmentu_zero_tablicy_transformat = (blockIdx.z * ROZMIAR_LATKI * ROZMIAR_LATKI * mnoznik_tablicy_transormat);
    float ilosc_niewyzerowanych = tablica_ilosci_zerowan[blockIdx.z];
    if ((row_pos < ROZMIAR_LATKI) && (col_pos < ROZMIAR_LATKI))
    {
        //sz_Macierz_wspolczynnikow_Kaizerra[threadIdx.y * ROZMIAR_LATKI + threadIdx.x] = Macierz_wspolczynnikow_Kaizerra[threadIdx.y * ROZMIAR_LATKI + threadIdx.x];
        __syncthreads();
        
        for (int i = 0; i < device_tablica_ilosci_pasujacych_latek[blockIdx.z]; i++)//�atka po �atce przepsisujemy �atki z  tablicy transformat device_tablice_transformaty_32_1krok(dla 1 kroku, dl 2 kroku device_tablice_transformaty_32 do t
        {
            int indeks_pomocniczy1_odkladanie_latek = index_2d_latki + (i * ROZMIAR_LATKI * ROZMIAR_LATKI);
            int indeks_pomocniczy2_odkladanie_latek = ((dev_koordynatySOA.koordynata_x[index_elmentu_zero_tablicy_koordynat + i] + index_x_pixela_gorny_lewy_obszaru_przeszukania) + col_pos) + ((dev_koordynatySOA.koordynata_y[index_elmentu_zero_tablicy_koordynat + i] + index_y_pixela_gorny_lewy_obszaru_przeszukania + row_pos) * szerokosc);
            obrazek_po_kolejnym_kroku[indeks_pomocniczy2_odkladanie_latek] += (Macierz_wspolczynnikow_Kaizerra[index_2d_latki] * ((device_tablice_transformaty_32_1krok[index_elmentu_zero_tablicy_transformat + indeks_pomocniczy1_odkladanie_latek]) / ilosc_niewyzerowanych));
            obrazek_po_kolejnym_kroku_dzielnik[indeks_pomocniczy2_odkladanie_latek] += (Macierz_wspolczynnikow_Kaizerra[index_2d_latki] / (ilosc_niewyzerowanych));
            __syncthreads();
        }
        __syncthreads();
    }
}

__global__ void Przepisywacz_z_tablic_transformaty_2krok(float* device_tablica_wartosci_fitru_wiena, float* obrazek_po_kolejnym_kroku, float* obrazek_po_kolejnym_kroku_dzielnik, int sigma, Tablice_koordynatLatek dev_koordynatySOA, int* device_tablica_ilosci_pasujacych_latek, float* device_tablice_transformaty_32_2krok, int ilosc_blokow_w_boku_x, int ilosc_blokow_w_boku_y, int szerokosc, int i, int j, int mnoznik_tablicy_transormat)
{
    int row_pos = threadIdx.y;
    int col_pos = threadIdx.x;
    int index_x_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z % ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + i;
    int index_y_pixela_gorny_lewy_obszaru_przeszukania = (blockIdx.z / ilosc_blokow_w_boku_x) * RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA + j;
    int index_2d_latki = col_pos + (row_pos * ROZMIAR_LATKI);
    int index_elmentu_zero_tablicy_koordynat = (blockIdx.z * ROZMIAR_OBSZARU_PRZESZUKANIA * ROZMIAR_OBSZARU_PRZESZUKANIA);
    int index_elmentu_zero_tablicy_transformat = (blockIdx.z * ROZMIAR_LATKI * ROZMIAR_LATKI * mnoznik_tablicy_transormat);
    if ((row_pos < ROZMIAR_LATKI) && (col_pos < ROZMIAR_LATKI))
    {
        for (int i = 0; i < device_tablica_ilosci_pasujacych_latek[blockIdx.z]; i++)//�atka po �atce przepsisujemy �atki z  tablicy transformat device_tablice_transformaty_32_1krok(dla 1 kroku, dl 2 kroku device_tablice_transformaty_32 do t
        {
            int indeks_pomocniczy1_odkladanie_latek = index_2d_latki + (i * ROZMIAR_LATKI * ROZMIAR_LATKI);
            int indeks_pomocniczy2_odkladanie_latek = ((dev_koordynatySOA.koordynata_x[index_elmentu_zero_tablicy_koordynat + i] + index_x_pixela_gorny_lewy_obszaru_przeszukania) + col_pos) + ((dev_koordynatySOA.koordynata_y[index_elmentu_zero_tablicy_koordynat + i] + index_y_pixela_gorny_lewy_obszaru_przeszukania + row_pos) * szerokosc);
            obrazek_po_kolejnym_kroku[indeks_pomocniczy2_odkladanie_latek] += (Macierz_wspolczynnikow_Kaizerra[index_2d_latki] * ((device_tablice_transformaty_32_2krok[index_elmentu_zero_tablicy_transformat + indeks_pomocniczy1_odkladanie_latek])) / device_tablica_ilosci_pasujacych_latek[blockIdx.z]);
            obrazek_po_kolejnym_kroku_dzielnik[indeks_pomocniczy2_odkladanie_latek] += (Macierz_wspolczynnikow_Kaizerra[index_2d_latki] / device_tablica_ilosci_pasujacych_latek[blockIdx.z]);
            __syncthreads();
        }
        __syncthreads();
    }
}


__global__ void Zerowanie(float* device_tablice_transformaty_32_1krok, int* device_tablica_ilosci_zerowan, int* device_tablica_ilosci_pasujacych_latek, float sigma, int mnoznik_tablicy_transormat)
{
    device_tablica_ilosci_zerowan[blockIdx.z] = 0;
    int indeks = blockIdx.x * blockDim.x + threadIdx.x;
    int przesuniecie = blockIdx.z * ROZMIAR_LATKI * ROZMIAR_LATKI * mnoznik_tablicy_transormat;
    int ilosc_pasujacych_latek = device_tablica_ilosci_pasujacych_latek[blockIdx.z];
    if (indeks < 1)
    {
        device_tablica_ilosci_zerowan[blockIdx.z] = 0;
    }
    __syncthreads();

    if (indeks < ilosc_pasujacych_latek * ROZMIAR_LATKI * ROZMIAR_LATKI)
    {
        if (abs(device_tablice_transformaty_32_1krok[indeks + przesuniecie]) < (LAMBDA3D_HARD * sigma))

        {
            device_tablice_transformaty_32_1krok[indeks + przesuniecie] = 0.0f;
        }
        else
        {
            atomicAdd(&device_tablica_ilosci_zerowan[blockIdx.z], 1);
        }
    }
    
}

__global__ void Filtr_Wiena(float* device_tablica_wartosci_fitru_wiena, float* device_tablice_transformaty_32_1krok, float* device_tablice_transformaty_32_2krok, int* device_tablica_ilosci_pasujacych_latek, float sigma, int mnoznik_tablicy_transormat)
{
    int przesuniecie = blockIdx.z * ROZMIAR_LATKI * ROZMIAR_LATKI * mnoznik_tablicy_transormat;
    float coef = 1.0f / (float)device_tablica_ilosci_pasujacych_latek[blockIdx.z];
    int indeks1 = blockIdx.x * blockDim.x + threadIdx.x;
    int indeks = blockIdx.x * blockDim.x + threadIdx.x + przesuniecie;
    if (indeks1 < 1)
    {
        device_tablica_wartosci_fitru_wiena[blockIdx.z] = 0.0;
    }
    //__syncthreads();

    if (device_tablica_ilosci_pasujacych_latek[blockIdx.z] > 0)
    {
        if (indeks1 < device_tablica_ilosci_pasujacych_latek[blockIdx.z] * ROZMIAR_LATKI * ROZMIAR_LATKI)
        {
            float x; //wartosc posrednia obliczen wspolczynnika filtracji wiena


            x = (device_tablice_transformaty_32_2krok[indeks] * device_tablice_transformaty_32_2krok[indeks]);
            float wspolczynnik_filtracji_wiena = x / (x + (float)(sigma * sigma));
            device_tablice_transformaty_32_2krok[indeks] = ((device_tablice_transformaty_32_1krok[indeks] * wspolczynnik_filtracji_wiena));
            //__syncthreads();
            //device_tablice_transformaty_32_1krok[indeks] = wspolczynnik_filtracji_wiena;
        }
    }

    //__syncthreads();

    if (indeks1 < 1)
    {
        device_tablica_wartosci_fitru_wiena[blockIdx.z] = 1.0 / device_tablica_ilosci_pasujacych_latek[blockIdx.z];
    }

    /*
   if (indeks1 < device_tablica_ilosci_pasujacych_latek[blockIdx.z] * ROZMIAR_LATKI * ROZMIAR_LATKI)
      {
      atomicAdd(&device_tablica_wartosci_fitru_wiena[blockIdx.z], device_tablice_transformaty_32_1krok[indeks]);
     }

   __syncthreads();

    if (indeks1 < 1)
    {
         if (device_tablica_wartosci_fitru_wiena[blockIdx.z] > 1.0f)
        {
             device_tablica_wartosci_fitru_wiena[blockIdx.z] = (1.0 / device_tablica_wartosci_fitru_wiena[blockIdx.z]) * coef;
             //device_tablica_wartosci_fitru_wiena[blockIdx.z] =  (1.0/ device_tablica_wartosci_fitru_wiena[blockIdx.z] * device_tablica_ilosci_pasujacych_latek[blockIdx.z]);
        }
        else
        {
            device_tablica_wartosci_fitru_wiena[blockIdx.z] = 1;
        }
        if (device_tablica_ilosci_pasujacych_latek[blockIdx.z] == 1)
        {
            device_tablica_wartosci_fitru_wiena[blockIdx.z] = 1.0 / (device_tablica_wartosci_fitru_wiena[blockIdx.z] * 32);
        }

    }
    */

}

__global__ void Nadpisywanie_marginesow1(float* device_obrazek_po1kroku, float* device_obrazek_po1kroku_dzielnik, int szerokosc, int wysokosc, int margines_lewy, int margines_prawy) //doanaie nowych margines�w
{

    int row_pos = blockIdx.y * blockDim.y + threadIdx.y;
    int col_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int margines = margines_lewy + margines_prawy;

    if (row_pos < szerokosc && col_pos < margines_lewy)

    {
        device_obrazek_po1kroku[col_pos + row_pos * szerokosc] = device_obrazek_po1kroku[(margines_lewy + (margines_lewy - col_pos)) + (row_pos * szerokosc)];
    }

    if (row_pos < wysokosc && col_pos < szerokosc)
    {
        if (col_pos > (szerokosc - margines_prawy))

        {
            device_obrazek_po1kroku[col_pos + row_pos * szerokosc] = device_obrazek_po1kroku[(szerokosc - margines_prawy - (col_pos - (szerokosc - margines_prawy))) + (row_pos * szerokosc)]; //-szerokosc + margines            
        }
    }

}

__global__ void Nadpisywanie_marginesow2(float* device_obrazek_po1kroku, float* device_obrazek_po1kroku_dzielnik, int szerokosc, int wysokosc, int margines_lewy, int margines_prawy) //dzielenie wyiku sumowania zerowanych �atek zprzez ilosc zerowa� oraz doanaie nowych margines�w
{

    int row_pos = blockIdx.y * blockDim.y + threadIdx.y;
    int col_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int margines = margines_lewy + margines_prawy;

    if (row_pos < margines_lewy && col_pos < szerokosc)

    {
        float x = device_obrazek_po1kroku[col_pos + (margines_lewy + (margines_lewy - row_pos)) * szerokosc];
        device_obrazek_po1kroku[col_pos + row_pos * szerokosc] = x;
    }

    if (row_pos < wysokosc && col_pos < szerokosc)
    {
        if (row_pos > wysokosc - margines_prawy)

        {
            device_obrazek_po1kroku[col_pos + row_pos * szerokosc] = device_obrazek_po1kroku[(col_pos + (wysokosc - margines_prawy - (row_pos - (wysokosc - margines_prawy))) * szerokosc)];

        }
    }
}



__global__ void DzielenieMacierzy(float* device_obrazek_po_n_kroku, float* __restrict__ device_obrazek_po_n_kroku_dzielnik, int szerokosc, int wysokosc, int margines_lewy, int margines_prawy) //dzielenie wyiku sumowania zerowanych �atek zprzez ilosc zerowa�
{

    int row_pos = blockIdx.y * blockDim.y + threadIdx.y;
    int col_pos = blockIdx.x * blockDim.x + threadIdx.x;


    int index2d_pixela = col_pos + row_pos * szerokosc;
    int margines = margines_lewy + margines_prawy;

    if (row_pos < wysokosc && col_pos < szerokosc)
    {
        device_obrazek_po_n_kroku[index2d_pixela] = device_obrazek_po_n_kroku[index2d_pixela] / device_obrazek_po_n_kroku_dzielnik[index2d_pixela];
    }
}

void initializeCUDA(int argc, char** argv, int& devID)
{
    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    hipError_t error;
    devID = 0;
    // get number of SMs on this GPU
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipDeviceProp_t deviceProp;

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
}

void dodanie_szumu(cv::Mat obrazek_zaszumiony, float sigm, int ilosc_kanalow)
{
    double sigma = sigm; // Warto�� sigma dla szumu gaussowskiego
    // Generator liczb losowych dla szumu gaussowskiego
    std::default_random_engine generator;
    std::normal_distribution<double> distribution(0.0, sigma);
    // Dodaje szum gaussowski do ka�dego piksela
    for (int y = 0; y < obrazek_zaszumiony.rows; y++)
    {
        for (int x = 0; x < obrazek_zaszumiony.cols; x++)
        {
            if (ilosc_kanalow == 1)
            {
                cv::Vec<uchar, 1>& pixele = obrazek_zaszumiony.at<cv::Vec<uchar, 1>>(y, x);
                for (int c = 0; c < 1; c++)
                {
                    double szum = distribution(generator);
                    int new_value = cv::saturate_cast<uchar>(pixele[c] + szum);
                    pixele[c] = new_value;
                }
            }
            else
            {
                cv::Vec3b& pixele = obrazek_zaszumiony.at<cv::Vec3b>(y, x);
                for (int c = 0; c < 3; c++)
                {
                    double szum = distribution(generator);
                    int new_value = cv::saturate_cast<uchar>(pixele[c] + szum);
                    pixele[c] = new_value;
                }
            }
        }
    }
}

void funkcja_glowna (cv::Mat Obrazek_zaszumiony, cv::Mat &Obrazek_odszumiony, float sigma, int szybkosc)
{
    int p_hard = P_HARD; //przesyni�cie pomidzy �atkami w kroku 1, w oryginale 1,2 lub 3, u Lebruna wynosi 3, w orginalnym opisie maksymalnie 4
    int p_wien = P_WIEN;//przesuni�cie pomi�dzy �atkami w kroku 2, w oryginale 1,2 lub 3, u Lebruna wynosi 3, w orginalnym opisie maksymalnie 4
    int zakladka_obszaru_przeszukania = 2;
    if (szybkosc == 0)
    {
        p_hard = P_HARD; //przesyni�cie pomidzy �atkami w kroku 1, w oryginale 1,2 lub 3, u Lebruna wynosi 3, w orginalnym opisie maksymalnie 4
        p_wien = P_WIEN;//przesuni�cie pomi�dzy �atkami w kroku 2, w oryginale 1,2 lub 3, u Lebruna wynosi 3, w orginalnym opisie maksymalnie 4
        zakladka_obszaru_przeszukania = 2;
    }
    else if (szybkosc == 1)
    {
        p_hard = 4;
        p_wien = 4;
        zakladka_obszaru_przeszukania = 5;
    }


    /////////////////////////////////////////////////////////////////////////////


    int szerokosc_obrazka_oryginalnego = Obrazek_zaszumiony.cols;
    int wysokosc_obrazka_oryginalnego = Obrazek_zaszumiony.rows;
    int wielkosc_marginesu_lewego = RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA / 2;
    int wielkosc_marginesu_prawego = RZECZYWISTY_ROZMIAR_OBSZARU_PRZESZUKANIA;

    cv::copyMakeBorder(Obrazek_zaszumiony, Obrazek_zaszumiony, wielkosc_marginesu_lewego, wielkosc_marginesu_prawego, wielkosc_marginesu_lewego, wielkosc_marginesu_prawego, cv::BORDER_REFLECT_101);
    Obrazek_zaszumiony.convertTo(Obrazek_zaszumiony, CV_32F);
    int szerokosc_obrazka_z_marginesami = szerokosc_obrazka_oryginalnego + wielkosc_marginesu_lewego + wielkosc_marginesu_prawego;
    int wysokosc_obrazka_z_marginesami = wysokosc_obrazka_oryginalnego + wielkosc_marginesu_lewego + wielkosc_marginesu_prawego;
    int wielkosc_tablicy_z_marginesami = szerokosc_obrazka_z_marginesami * wysokosc_obrazka_z_marginesami;

    Obrazek_zaszumiony = Obrazek_zaszumiony.reshape(1, wielkosc_tablicy_z_marginesami);
    //////////////////////////////////////////////////////////////////tworzymy zmienne do przekazania do Kernelu starowego///////////////////////////

    float* host_obrazek_poczatkowy = new float[wielkosc_tablicy_z_marginesami];
    float* host_obrazek_po1kroku = new float[wielkosc_tablicy_z_marginesami];
    float* host_obrazek_po2kroku = new float[wielkosc_tablicy_z_marginesami];
    float* host_obrazek_po1kroku_dzielnik = new float[wielkosc_tablicy_z_marginesami];

    int ilosc_blokow_w_boku_x = (int)std::ceil(((double)szerokosc_obrazka_oryginalnego / (ROZMIAR_OBSZARU_PRZESZUKANIA + ROZMIAR_LATKI)));
    int ilosc_blokow_w_boku_y = (int)ceil(((double)wysokosc_obrazka_oryginalnego) / (ROZMIAR_OBSZARU_PRZESZUKANIA + ROZMIAR_LATKI));
    int ilosc_blokow = ilosc_blokow_w_boku_x * ilosc_blokow_w_boku_y;

    int wielkosc_tablicy_transformaty_32 = ilosc_blokow * ROZMIAR_LATKI * ROZMIAR_LATKI * N_WIEN;
    int wielkosc_tablicy_transformaty_16 = ilosc_blokow * ROZMIAR_LATKI * ROZMIAR_LATKI * N_HARD;
    int wielkosc_tablicy_ilosci_pasujacych_latek = ilosc_blokow;
    int wielkosc_tablicy_zerowan = ilosc_blokow;
    int wielkosc_tablicy_koordynat = ilosc_blokow * ROZMIAR_OBSZARU_PRZESZUKANIA * ROZMIAR_OBSZARU_PRZESZUKANIA;


    host_obrazek_poczatkowy = (float*)Obrazek_zaszumiony.data;

    float* device_obrazek_poczatkowy;
    float* device_obrazek_po1kroku;
    float* device_obrazek_po2kroku;
    float* device_obrazek_po1kroku_dzielnik;
    float* device_obrazek_po2kroku_dzielnik;
    int start_1 = clock();
    //hipFree(0);
    hipMalloc((void**)&device_obrazek_poczatkowy, wielkosc_tablicy_z_marginesami * sizeof(float));
    int stop_1 = clock();
    hipMalloc((void**)&device_obrazek_po1kroku, wielkosc_tablicy_z_marginesami * sizeof(float));
    hipMalloc((void**)&device_obrazek_po2kroku, wielkosc_tablicy_z_marginesami * sizeof(float));
    hipMalloc((void**)&device_obrazek_po1kroku_dzielnik, wielkosc_tablicy_z_marginesami * sizeof(float));
    hipMalloc((void**)&device_obrazek_po2kroku_dzielnik, wielkosc_tablicy_z_marginesami * sizeof(float));

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    ////////////////////////////////////////////przygotowanie i lokowanie w pami�ci tablic pomocniczych////////////////////////////////
    /////////lokujemy je w pami�ci przed rozpocz�ciem wykonywania programu przez kart� gdy� dynamiczne lokowanie pami�ci przez CUD� wielokrotnie spowalnia program///////////////


    int rozmiar_w_pamieci_tablic_koordynat_inty = sizeof(int) * wielkosc_tablicy_koordynat;
    int rozmiar_w_pamieci_tablic_koordynat_floaty = sizeof(float) * wielkosc_tablicy_koordynat;
    Tablice_koordynatLatek dev_koordynatySOA;
    Tablice_koordynatLatek* wskaznik_dev_koordynatySOA = &dev_koordynatySOA;


    hipMalloc((&wskaznik_dev_koordynatySOA->MSE), rozmiar_w_pamieci_tablic_koordynat_floaty);
    hipMalloc(&(wskaznik_dev_koordynatySOA->koordynata_x), rozmiar_w_pamieci_tablic_koordynat_inty);
    hipMalloc(&(wskaznik_dev_koordynatySOA->koordynata_y), rozmiar_w_pamieci_tablic_koordynat_inty);

    float* device_tablice_transformaty_16;
    float* device_tablice_transformaty_32_1krok;
    float* device_tablice_transformaty_32_2krok;
    int* device_tablica_ilosci_pasujacych_latek;
    //int* device_tablica_do_najmniejszych;
    int* device_tablica_ilosci_zerowan;
    float* device_tablica_wartosci_fitru_wiena;
    hipMalloc((void**)&device_tablice_transformaty_16, wielkosc_tablicy_transformaty_16 * sizeof(float));
    hipMalloc((void**)&device_tablice_transformaty_32_1krok, wielkosc_tablicy_transformaty_32 * sizeof(float));
    hipMalloc((void**)&device_tablice_transformaty_32_2krok, wielkosc_tablicy_transformaty_32 * sizeof(float));
    hipMalloc(&device_tablica_ilosci_pasujacych_latek, wielkosc_tablicy_ilosci_pasujacych_latek * sizeof(int));
    hipMalloc((void**)&device_tablica_ilosci_zerowan, wielkosc_tablicy_zerowan * sizeof(int));
    hipMalloc((void**)&device_tablica_wartosci_fitru_wiena, wielkosc_tablicy_zerowan * sizeof(float));
    //hipMalloc(&device_tablica_do_najmniejszych, ilosc_blokow_w_bloku_x * ilosc_blokow_w_bloku_y * N_WIEN * sizeof(int));
    int start = clock();
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(Kalkulator_MSE_szum_duzy), hipFuncCachePreferShared);
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(DCT), hipFuncCachePreferShared);
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(DCT_odwrotna), hipFuncCachePreferShared);
   /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    int start_3 = clock();
    hipMemcpy(device_obrazek_poczatkowy, host_obrazek_poczatkowy, wielkosc_tablicy_z_marginesami * sizeof(float), hipMemcpyHostToDevice);
    int x = 11;
    int ilosc_latek_w_obszarze_przeszukania = (ROZMIAR_OBSZARU_PRZESZUKANIA) * (ROZMIAR_OBSZARU_PRZESZUKANIA);
    dim3 bloki_Kalkulator_MSE(1, 1, ilosc_blokow);
    dim3 watki_Kalkulator_MSE(ROZMIAR_OBSZARU_PRZESZUKANIA, ROZMIAR_OBSZARU_PRZESZUKANIA, 1);
    dim3 bloki_Kalkulator_MSE_szum_duzy(ROZMIAR_OBSZARU_PRZESZUKANIA, ROZMIAR_OBSZARU_PRZESZUKANIA, ilosc_blokow);
    dim3 watki_Kalkulator_MSE_szum_duzy(ROZMIAR_LATKI, ROZMIAR_LATKI, 1);
    dim3 bloki_najmniejsze_liczby(1, 1, ilosc_blokow);
    dim3 watki_najmniejsze_liczby(ilosc_latek_w_obszarze_przeszukania / 2, 1, 1);
    dim3 watki_Przepisywacz(ROZMIAR_LATKI, ROZMIAR_LATKI, 1);
    dim3 bloki_Przepisywacz(1, 1, ilosc_blokow);
    dim3 bloki_DCT_krok1(1, 1, ilosc_blokow * N_HARD);
    dim3 watki_DCT_krok1(ROZMIAR_LATKI, ROZMIAR_LATKI, 1);
    dim3 bloki_DCT_krok2(1, 1, ilosc_blokow * N_WIEN);
    dim3 watki_DCT_krok2(ROZMIAR_LATKI, ROZMIAR_LATKI, 1);
    dim3 bloki_Walsh(1, 1, ilosc_blokow);
    dim3 watki_Walsh_krok1(ROZMIAR_LATKI * ROZMIAR_LATKI, N_HARD / 2, 1);
    dim3 watki_Walsh_krok2(ROZMIAR_LATKI * ROZMIAR_LATKI, N_WIEN / 2, 1);

    dim3 bloki_Zerowanie(1, 1, ilosc_blokow);
    dim3 watki_Zerowanie(N_HARD * ROZMIAR_LATKI * ROZMIAR_LATKI, 1, 1);
    dim3 bloki_Wien(N_WIEN, 1, ilosc_blokow);
    dim3 watki_Wien(ROZMIAR_LATKI * ROZMIAR_LATKI, 1, 1);
    int dzielenie_macierzy_watki_x = 32;
    int dzielenie_macierzy_watki_y = 32;
    int dzielenie_macierzy_bloki_x = (szerokosc_obrazka_z_marginesami + dzielenie_macierzy_watki_x) / dzielenie_macierzy_watki_x;
    int dzielenie_macierzy_bloki_y = (wysokosc_obrazka_z_marginesami + dzielenie_macierzy_watki_y) / dzielenie_macierzy_watki_y;
    dim3 bloki_dzielnie_Macierzy(dzielenie_macierzy_bloki_x, dzielenie_macierzy_bloki_y, 1);
    dim3 watki_dzielnie_Macierzy(dzielenie_macierzy_watki_x, dzielenie_macierzy_watki_y, 1);
    for (int i = 0; i < (ROZMIAR_OBSZARU_PRZESZUKANIA + zakladka_obszaru_przeszukania); i += p_hard)
    {
        for (int j = 0; j < (ROZMIAR_OBSZARU_PRZESZUKANIA + zakladka_obszaru_przeszukania); j += p_hard)
        {
            //int wielkosc_tabeli_koordynat = (ROZMIAR_OBSZARU_PRZESZUKANIA) * (ROZMIAR_OBSZARU_PRZESZUKANIA);
            int szerokosc = szerokosc_obrazka_z_marginesami;
            int tau_hard = TAU_HARD_NISKI;
            if (sigma > 40)
            {
                tau_hard = TAU_HARD_WYSOKI * 3;

                Kalkulator_MSE_szum_duzy << <bloki_Kalkulator_MSE_szum_duzy, watki_Kalkulator_MSE_szum_duzy >> > (device_obrazek_poczatkowy, dev_koordynatySOA, ilosc_blokow_w_boku_x, szerokosc, i, j, sigma);

                //Kalkulator_MSE << <bloki_Kalkulator_MSE, watki_Kalkulator_MSE >> > (device_obrazek_poczatkowy, dev_koordynatySOA, ilosc_blokow_w_boku_x, ilosc_blokow_w_boku_y, szerokosc, i, j);

            }
            else
            {
                Kalkulator_MSE << <bloki_Kalkulator_MSE, watki_Kalkulator_MSE >> > (device_obrazek_poczatkowy, dev_koordynatySOA, ilosc_blokow_w_boku_x, ilosc_blokow_w_boku_y, szerokosc, i, j);
            }

            ///////////////////////////////////////////////////////////wyszukanie N_HARD najblizszych latek////////////////////////////////////////////

            Najmniejsze_liczby << <bloki_najmniejsze_liczby, watki_najmniejsze_liczby >> > (dev_koordynatySOA, device_tablica_ilosci_pasujacych_latek, N_HARD, tau_hard, false);
            Przepisywacz_do_tabloc_transformaty << <bloki_Przepisywacz, watki_Przepisywacz >> > (device_obrazek_poczatkowy, dev_koordynatySOA, device_tablica_ilosci_pasujacych_latek, device_tablice_transformaty_16, ilosc_blokow_w_boku_x, 0, szerokosc, i, j, N_HARD);

            ////////////////////////////////////////////// pasuj�ce �atki znajduj�ce si� w "device_tablice_transformaty_32_1krok" (tylko tyle z tej tablicy ile spe�nia warunek max dopasowania) poddajemy transformacie cosinusowej 2d (ca�e �atki), a nast�pnie transformacie 1D walsha-hadamarda "w poprzek" grupy �atek////////////////////////////////////////
            DCT << <bloki_DCT_krok1, watki_DCT_krok1 >> > (device_tablice_transformaty_16, device_tablice_transformaty_32_2krok, ROZMIAR_LATKI, ROZMIAR_LATKI, device_tablica_ilosci_pasujacych_latek, N_HARD, false);
            Walsh1dPojedyncza << <bloki_Walsh, watki_Walsh_krok1 >> > (device_tablice_transformaty_16, device_tablice_transformaty_32_2krok, device_tablica_ilosci_pasujacych_latek, N_HARD, false);
            Zerowanie << <bloki_Zerowanie, watki_Zerowanie >> > (device_tablice_transformaty_16, device_tablica_ilosci_zerowan, device_tablica_ilosci_pasujacych_latek, sigma, N_HARD);

            ////////////////////////////////////////////////////////////// Odwracamy transformaty w celu uzyskania w�a�ciwego obrazu//////////////////////////////////////////////////////

            Walsh1dPojedyncza << <bloki_Walsh, watki_Walsh_krok1 >> > (device_tablice_transformaty_16, device_tablice_transformaty_32_2krok, device_tablica_ilosci_pasujacych_latek, N_HARD, false);
            DCT_odwrotna << <bloki_DCT_krok1, watki_DCT_krok1 >> > (device_tablice_transformaty_16, ROZMIAR_LATKI, ROZMIAR_LATKI, device_tablica_ilosci_pasujacych_latek, N_HARD);
            ///// //////////////////////////teraz trzeba poodk�ada� l��tki w odpowiednie miejsca tablicy wynikowej po 1 kroku, oraz pododawa� warto�ci ili�ci niewyzerowanych w jej dzielniku
            Przepisywacz_z_tablic_transformaty_1krok << <bloki_Przepisywacz, watki_Przepisywacz >> > (device_tablica_ilosci_zerowan, device_obrazek_po1kroku, device_obrazek_po1kroku_dzielnik, dev_koordynatySOA, device_tablica_ilosci_pasujacych_latek, device_tablice_transformaty_16, ilosc_blokow_w_boku_x, ilosc_blokow_w_boku_y, szerokosc, i, j, N_HARD);
        }
    }

    DzielenieMacierzy << <bloki_dzielnie_Macierzy, watki_dzielnie_Macierzy >> > (device_obrazek_po1kroku, device_obrazek_po1kroku_dzielnik, szerokosc_obrazka_z_marginesami, wysokosc_obrazka_z_marginesami, wielkosc_marginesu_lewego, wielkosc_marginesu_prawego);

    Nadpisywanie_marginesow1 << <bloki_dzielnie_Macierzy, watki_dzielnie_Macierzy >> > (device_obrazek_po1kroku, device_obrazek_po1kroku_dzielnik, szerokosc_obrazka_z_marginesami, wysokosc_obrazka_z_marginesami, wielkosc_marginesu_lewego, wielkosc_marginesu_prawego);
    Nadpisywanie_marginesow2 << <bloki_dzielnie_Macierzy, watki_dzielnie_Macierzy >> > (device_obrazek_po1kroku, device_obrazek_po1kroku_dzielnik, szerokosc_obrazka_z_marginesami, wysokosc_obrazka_z_marginesami, wielkosc_marginesu_lewego, wielkosc_marginesu_prawego);
    for (int i = 0; i < (ROZMIAR_OBSZARU_PRZESZUKANIA + zakladka_obszaru_przeszukania); i += p_wien)
    {
        for (int j = 0; j < (ROZMIAR_OBSZARU_PRZESZUKANIA + zakladka_obszaru_przeszukania); j += p_wien)
        {

            //int wielkosc_tabeli_koordynat = (ROZMIAR_OBSZARU_PRZESZUKANIA) * (ROZMIAR_OBSZARU_PRZESZUKANIA);
            int szerokosc = szerokosc_obrazka_z_marginesami;
            int tau_wien;
            if (sigma < 40)
            {
                tau_wien = 1000;
            }
            else
                tau_wien = 3000;

            Kalkulator_MSE << <bloki_Kalkulator_MSE, watki_Kalkulator_MSE >> > (device_obrazek_po1kroku, dev_koordynatySOA, ilosc_blokow_w_boku_x, ilosc_blokow_w_boku_y, szerokosc, i, j);

            ///////////////////////////////////////////////////////////wyszukanie N_WIEN najblizszych latek////////////////////////////////////////////

            Najmniejsze_liczby << <bloki_najmniejsze_liczby, watki_najmniejsze_liczby >> > (dev_koordynatySOA, device_tablica_ilosci_pasujacych_latek, N_WIEN, tau_wien, true);
            //przepisujemy �atki z tablicy reprezentuj�cej obrazek wej�ciowego do "device_tablice_transformaty_32_1krok": 
            Przepisywacz_do_tabloc_transformaty << <bloki_Przepisywacz, watki_Przepisywacz >> > (device_obrazek_poczatkowy, dev_koordynatySOA, device_tablica_ilosci_pasujacych_latek, device_tablice_transformaty_32_1krok, ilosc_blokow_w_boku_x, ilosc_blokow_w_boku_y, szerokosc, i, j, N_WIEN);
            //przepisujemy �atki z tablicy repezentuj�cej obrazek wst�pnie odszumiony w 1 kroku do device_tablice_transformaty_32_2krok
            Przepisywacz_do_tabloc_transformaty << <bloki_Przepisywacz, watki_Przepisywacz >> > (device_obrazek_po1kroku, dev_koordynatySOA, device_tablica_ilosci_pasujacych_latek, device_tablice_transformaty_32_2krok, ilosc_blokow_w_boku_x, ilosc_blokow_w_boku_y, szerokosc, i, j, N_WIEN);
            ////////////////////////////////////////////// pasuj�ce �atki znajduj�ce si� w "device_tablice_transformaty_32_1krok" (tylko tyle z tej tablicy ile spe�nia warunek max dopasowania) poddajemy transformacie cosinusowej 2d (ca�e �atki), a nast�pnie transformacie 1D walsha-hadamarda "w poprzek" grupy �atek////////////////////////////////////////
            DCT << <bloki_DCT_krok2, watki_DCT_krok2 >> > (device_tablice_transformaty_32_1krok, device_tablice_transformaty_32_2krok, ROZMIAR_LATKI, ROZMIAR_LATKI, device_tablica_ilosci_pasujacych_latek, N_WIEN, true);
            Walsh1dPojedyncza << <bloki_Walsh, watki_Walsh_krok2 >> > (device_tablice_transformaty_32_1krok, device_tablice_transformaty_32_2krok, device_tablica_ilosci_pasujacych_latek, N_WIEN, true); // przesuni�cie to indeks elentu zerowego w macierzy transformat dla danego wywo�ania kernela                                                                                                                      
            /////////////////////////////////////////////////////////////// W przekszta�conych �atkach zerujemy wsp�czynniki kt�rych abs jest mmniejszy ni� Lambda_Hard_3D*SIGMA/////////////////////////

            Filtr_Wiena << <bloki_Wien, watki_Wien >> > (device_tablica_wartosci_fitru_wiena, device_tablice_transformaty_32_1krok, device_tablice_transformaty_32_2krok, device_tablica_ilosci_pasujacych_latek, sigma, N_WIEN);
            Walsh1dPojedyncza << <bloki_Walsh, watki_Walsh_krok2 >> > (device_tablice_transformaty_32_2krok, device_tablice_transformaty_32_1krok, device_tablica_ilosci_pasujacych_latek, N_WIEN, false);
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(DCT_odwrotna), hipFuncCachePreferShared);
            DCT_odwrotna << <bloki_DCT_krok2, watki_DCT_krok2 >> > (device_tablice_transformaty_32_2krok, ROZMIAR_LATKI, ROZMIAR_LATKI, device_tablica_ilosci_pasujacych_latek, N_WIEN);

            Przepisywacz_z_tablic_transformaty_2krok << <bloki_Przepisywacz, watki_Przepisywacz >> > (device_tablica_wartosci_fitru_wiena, device_obrazek_po2kroku, device_obrazek_po2kroku_dzielnik, sigma, dev_koordynatySOA, device_tablica_ilosci_pasujacych_latek, device_tablice_transformaty_32_2krok, ilosc_blokow_w_boku_x, ilosc_blokow_w_boku_y, szerokosc, i, j, N_WIEN);
        }
    }


    DzielenieMacierzy << <bloki_dzielnie_Macierzy, watki_dzielnie_Macierzy >> > (device_obrazek_po2kroku, device_obrazek_po2kroku_dzielnik, szerokosc_obrazka_z_marginesami, wysokosc_obrazka_z_marginesami, wielkosc_marginesu_lewego, wielkosc_marginesu_prawego);
    hipMemcpy(host_obrazek_po2kroku, device_obrazek_po2kroku, wielkosc_tablicy_z_marginesami * sizeof(float), hipMemcpyDeviceToHost);

   cv::Mat testDataMat2(wysokosc_obrazka_z_marginesami, szerokosc_obrazka_z_marginesami, CV_32F, host_obrazek_po2kroku);
  
    testDataMat2.convertTo(testDataMat2, CV_8U);
    Obrazek_odszumiony = testDataMat2(cv::Rect(wielkosc_marginesu_lewego, wielkosc_marginesu_lewego, szerokosc_obrazka_oryginalnego, wysokosc_obrazka_oryginalnego));



    hipFree(device_obrazek_poczatkowy);
    hipFree(device_obrazek_po1kroku);
    hipFree(device_tablice_transformaty_32_1krok);
    hipFree(device_tablice_transformaty_16);
    hipFree(device_tablice_transformaty_32_2krok);
    hipFree(device_tablica_ilosci_pasujacych_latek);
    hipFree(device_tablica_ilosci_zerowan);
    hipFree(device_obrazek_po2kroku);
    hipFree(device_tablica_wartosci_fitru_wiena);
    hipFree(device_obrazek_po2kroku_dzielnik);
    hipFree(device_obrazek_po1kroku_dzielnik);

    hipFree(wskaznik_dev_koordynatySOA->MSE);
    hipFree(wskaznik_dev_koordynatySOA->koordynata_x);
    hipFree(wskaznik_dev_koordynatySOA->koordynata_y);
    wskaznik_dev_koordynatySOA->MSE = nullptr;
    wskaznik_dev_koordynatySOA->koordynata_x = nullptr;
    wskaznik_dev_koordynatySOA->koordynata_y = nullptr;
    //hipFree(&dev_koordynatySOA);
    hipFree(wskaznik_dev_koordynatySOA);
 

    delete[] host_obrazek_po2kroku;
;
}

int main(int argc, char** argv)
{
    float sigma = SIGMA;
    cv::Mat Obrazek;
    cv::Mat Obrazek_odszumiony;
    int tryb_szybkosci = 1;
    int opcja_obrazka=1;
    int licznik = 0;
    std::string wpisana_nazwa;
	
	if (argc == 2 && (std::string(argv[1]) == "--help" || std::string(argv[1]) == "-h"))
    {
        std::cout << "Filtr NLM GPU, filtruje z szumu obrazy w skali szarosci.\n";
        std::cout << "Uzycie: BM3D_Gray_GPU.exe <liczba calkowita>  <liczba calkowita lub zmiennoprzecinkowa>\n ";
        std::cout << "Argumenty:\n";
        std::cout << "  <nazwa pliku>         Nazwa pliku. Mozna podac nazwe i sciezke folderu lub sama nazwe \n";
        std::cout << "                        jezeli znajduje sie w jednym folderze z programem\n";
        std::cout << "                        -zostan� przetworzone wszystkie pliki graficzne w folderze\n";
        std::cout << "  <poziom szumu>        Liczba calkowita: 0 do 100\n";
        std::cout << "  <stala filtracji>	Liczba calkowita: sila odzumiania";
        return 0;
    }

    if (argc != 4) {
        std::cerr << "U�ycie: " << argv[0] << " <nazwa pliku> <poziom szumu> <normalny czy szybki?>\n pomoc: --help lub -h";
        cv::waitKey(0);
        return 1;
    }

    wpisana_nazwa = argv[1];       //nazwa wczytywanego pliku argv[0] to nazwa programu
    sigma = std::atoi(argv[2]);  // drugi arg - poziom szumu
    tryb_szybkosci = std::atoi(argv[3]); // Argument float

    int devID = 0;
    initializeCUDA(argc, argv, devID);
    hipDeviceReset();
    
    time_t czasStart = clock();

    if (std::filesystem::is_regular_file(wpisana_nazwa))
    {
        Obrazek = cv::imread(wpisana_nazwa, cv::IMREAD_GRAYSCALE);

        if (Obrazek.empty())
        {
            std::cerr << "Nie mozna wczytac obrazka do odszumienia." << std::endl;
            cv::waitKey(0);
            return -1;
        }
        funkcja_glowna(Obrazek, Obrazek_odszumiony, sigma, tryb_szybkosci);
        for (std::filesystem::path plik : {std::filesystem::absolute(wpisana_nazwa)})
        {
            std::string nowa_nazwa = plik.stem().string() + "_filtered" + plik.extension().string();
            std::string nowa_nazwa_i_sciezka = plik.parent_path().string() + "/" + nowa_nazwa;
            std::cout << plik.parent_path().string() << std::endl;
            std::cout << nowa_nazwa_i_sciezka << std::endl;
            // Zapis przetworzonego obrazu
            cv::imshow("Obrazek po 2 kroku", Obrazek_odszumiony);
            cv::imwrite(nowa_nazwa_i_sciezka, Obrazek_odszumiony);
        }
        licznik++;
    }
    else if (std::filesystem::is_directory(wpisana_nazwa)) {
        for (const auto& entry : std::filesystem::directory_iterator(wpisana_nazwa)) {
            // Sprawdzenie, czy plik ma odpowiednie rozszerzenie
            if (entry.is_regular_file() && (entry.path().extension() == ".jpg" || entry.path().extension() == ".jpeg" || entry.path().extension() == ".png"
                || entry.path().extension() == ".bmp" || entry.path().extension() == ".tiff" || entry.path().extension() == ".tif" || entry.path().extension() == ".webp"
                || entry.path().extension() == ".hdr" || entry.path().extension() == ".jp2"))
            {
                Obrazek = cv::imread(entry.path().string(), cv::IMREAD_GRAYSCALE);

                if (Obrazek.empty())
                {
                    std::cerr << "Nie mozna wczytac obrazka do odszumienia." << std::endl;
                    cv::waitKey(0);
                    return -1;
                }
                funkcja_glowna(Obrazek, Obrazek_odszumiony, sigma, tryb_szybkosci);
                for (std::filesystem::path plik : {std::filesystem::absolute(std::filesystem::path(entry))})
                {
                    std::string nowa_nazwa = plik.stem().string() + "_filtered" + plik.extension().string();
                    std::string nowa_sciezka = plik.parent_path().string() + "/" +"filtered";
                    std::string nowa_nazwa_i_sciezka = nowa_sciezka + "/" + nowa_nazwa;

                    if (!std::filesystem::exists(nowa_sciezka)) //sprawdza czy istnieje folder do apisania wynikowych obrazow
                    {
                        if (std::filesystem::create_directories(nowa_sciezka)) {
                            std::cout << "Utworzono folder: " << nowa_sciezka << std::endl;
                        }
                        else {
                            std::cerr << "Nie uda�o si� utworzy� folderu: " << nowa_sciezka << std::endl;
                            return 1;
                        }
                    }
                    cv::imwrite(nowa_nazwa_i_sciezka, Obrazek_odszumiony);
                    std::cerr << "przefiltrowano i zapisano:" << nowa_nazwa << std::endl;
                    licznik++;
                }

            }
        }
    }
    else {
        std::cerr << "Podana �cie�ka lub nazwa pliku  jest bledna" << std::endl;
        return 1;
    }
    time_t czasStop = clock();
    double czas = (double)(czasStop - czasStart) / (double)CLOCKS_PER_SEC;
    std::cout << "Przefiltrowano " << licznik << " obrazow w czasie : " << czas << " s." << std::endl;   
    cv::waitKey(0);

    hipDeviceReset();
    //getch();

    return 0;

}